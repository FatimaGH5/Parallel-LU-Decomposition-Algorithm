#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>
using namespace std;




//LU Decompistion


__global__ void first_for (float** a ,float** l,float** u, int size ,int i )
{
int row = blockIdx.x * blockDim.x + threadIdx.x;


if (row < size)
        {
            //if row is smaller than i, set l[j][i] to 0
            if (row < i)
            {
                l[row][i] = 0;
                
            }
            else
            {
            //otherwise, do some math to get the right value
            l[row][i] = a[row][i];
            for (int k = 0; k < i; k++)
            {
                //deduct from the current l cell the value of these 2 values multiplied
                l[row][i] = l[row][i] - l[row][k] * u[k][i];
            }
            }
        }

}


__global__ void second_for (float** a ,float** l,float** u, int size ,int i )
{
int col = blockIdx.x * blockDim.x + threadIdx.x;


if (col < size)
        {
            //if col is smaller than i, set u's current index to 0
            if (col < i)
            {
                u[i][col] = 0;
                
            }

            //if they're equal, set u's current index to 1
            else if (col == i)
            {
                u[i][col] = 1;
                
            }
            else
            {
            //otherwise, do some math to get the right value
            u[i][col] = a[i][col] / l[i][i];
            for (int k = 0; k < i; k++)
            {
                u[i][col] = u[i][col] - ((l[i][k] * u[k][col]) / l[i][i]);
            }
            }

        }

}






//print the matrix out
void print_matrix(float** matrix, int size)
{
    //for each row...
    for (int i = 0; i < size; i++)
    {
        //for each column
        for (int j = 0; j < size; j++)
        {
            //print out the cell
            cout << left << setw(9) << setprecision(3) << matrix[i][j] << left <<  setw(9);
        }
        //new line when ever row is done
        cout << endl;
    }
}





//fill the array with random values (done for a)
void random_fill(float** matrix, int size)
{
    //fill a with random values
    cout << "Producing random values " << endl;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            matrix[i][j] = ((rand()%10)+1) ;
        }
    }

    //Ensure the matrix is diagonal dominant to guarantee invertible-ness
    //diagCount well help keep track of which column the diagonal is in
    int diagCount = 0;
    float sum = 0;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            //Sum all column vaalues
            sum += abs(matrix[i][j]);
        }
        //Remove the diagonal  value from the sum
        sum -= abs(matrix[i][diagCount]);
        //Add a random value to the sum and place in diagonal position
        matrix[i][diagCount] = sum + ((rand()%5)+1);
        ++diagCount;
        sum = 0;
    }
}
  //allocate vectors on unified memory **

 void initialize_matrices(float** a, float** l, float** u, int size)
  {



  for(int i=0; i<size; i++){
    hipMallocManaged(&a[i], size * sizeof(float));
    hipMallocManaged(&l[i], size * sizeof(float));
    hipMallocManaged(&u[i], size * sizeof(float));

  }
}




int main(){
  int n=3000;

  float **a, **l, **u;
   int i =0;

  srand(1);
  hipMallocManaged(&a, n * sizeof(float*));
  hipMallocManaged(&l, n * sizeof(float*));
  hipMallocManaged(&u, n * sizeof(float*));
initialize_matrices(a,l,u,n);
random_fill(a, n);


// add the clock
 double runtime;
  runtime = clock()/(double)CLOCKS_PER_SEC;
for( i=0; i<n; ++i){


    first_for <<< 32,250 >>> (a , l, u, n , i );
    hipDeviceSynchronize();
    second_for <<< 32,250 >>> ( a , l, u, n , i );
    hipDeviceSynchronize();

   }

runtime = clock() - runtime;
cout << "Runtime for LU Decomposition is: " << (runtime)/(double)(CLOCKS_PER_SEC) << endl;






/*
// print the matrices
cout << "A Matrix: " << endl;
print_matrix(a, n);
cout << "L Matrix: " << endl;
print_matrix(l, n);

cout << "U Matrix: " << endl;
print_matrix(u, n);
*/
cout << "Runtime for LU Decomposition is: " << (runtime)/float(CLOCKS_PER_SEC) << endl;


for(i=0; i<n; i++){
    hipFree(a[i]);
     hipFree(l[i]);
     hipFree(u[i]);
  }
   hipFree(a);
     hipFree(l);
     hipFree(u);

  return 0;
}